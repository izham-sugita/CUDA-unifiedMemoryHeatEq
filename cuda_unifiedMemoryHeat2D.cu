
#include <hip/hip_runtime.h>
#include<iostream>
#include<iomanip>
#include<fstream>
#include<vector>
#include<utility>
#include<chrono>

#include<cstdlib>
#include<cstdio>
#include<cmath>

#define pi 4.0*atan(1.0)
#define blockDim_x 128
#define blockDim_y 8

using namespace std;
using namespace std::chrono;

/*
__global__ void add(int N, float *x, float *y){

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
  for(int i=index; i<N; i+=stride){
    y[i] = x[i] + y[i];
  }
}
*/

//monolithic kernel
__global__  void  cuda_diffusion2d_0
(
   float    *f,         /* dependent variable                        */
   float    *fn,        /* dependent variable                        */
   int      nx,         /* grid number in the x-direction            */
   int      ny,         /* grid number in the x-direction            */
   float    c0,         /* coefficient no.0                          */
   float    c1,         /* coefficient no.1                          */
   float    c2          /* coefficient no.2                          */
)
{
   int    j,    jx,   jy;
   float  fcc,  fce,  fcw,  fcs,  fcn;

   jy = blockDim.y*blockIdx.y + threadIdx.y;
   jx = blockDim.x*blockIdx.x + threadIdx.x;

   //Dirichilet BC
if(jx > 0  && jx < nx-1){
if(jy > 0 && jy < ny-1){
   j = nx*jy + jx;
   fcc = f[j];
   fcw = f[j - 1];
   fce = f[j+1];
   fcs = f[j-nx];
   fcn = f[j+nx];

   fn[j] = c0*(fce + fcw)
         + c1*(fcn + fcs)
         + c2*fcc;
 }
 }
   
}

int main()
{

  int nx;
  int ny;
  cout<<"Enter nx, ny "<<endl;
  cin>>nx;
  cin>>ny;
  
  float dx = 1.0/(float)(nx-1);
  float dy = 1.0/(float)(ny-1);
  float dt = 0.01*(dx*dx);

  //allocate arrays and initial condition
  //using unified memory
  float *Told, *Tnew;
  hipMallocManaged(&Told,(nx*ny)*sizeof(float));
  hipMallocManaged(&Tnew,(nx*ny)*sizeof(float));

  for(int i=0; i<nx; i++){
    for(int j=0; j<ny; j++){
      int id = i*ny + j;
      Told[id] = sin((float)i*dx*pi)*sin((float)j*dy*pi);
      Tnew[id] = 0.0f;
    }
  }

  float kappa = 1.0;
  float c0 = kappa*dt/(dx*dx),
        c1 = kappa*dt/(dy*dy),
        c2 = 1.0 - 2.0*(c0 + c1);
  
  int gridX = nx/blockDim_x;
  int gridY = ny/blockDim_y;

  //CUDA specific object type
  dim3 grid(gridX,gridY,1), threads(blockDim_x,blockDim_y,1);

  //time loop
  int iter = 0;
  int itermax = 20000;
  double operation = 0.0;

  high_resolution_clock::time_point t1 =
    high_resolution_clock::now();
  
  do{
  
  //run kernel on gpu
  cuda_diffusion2d_0<<<grid,threads>>>(Told,Tnew,nx,ny,c0,c1,c2);
  swap(Told,Tnew);

  if(iter%1000 == 0) cout<<"Step : "<<iter<<endl;

  operation += 7.0*(double)ny *(double)nx;
  iter +=1;
    
  }while(iter<itermax+1);

  high_resolution_clock::time_point t2 =
    high_resolution_clock::now();

  duration<double> elapsed_time = duration_cast<duration<double> >(t2-t1);

  cout<<"Operations : "<<operation<<endl;
  cout<<"Elapsed time : "<<elapsed_time.count()<<" secs."<<endl;
  double flops = operation /(elapsed_time.count()*1e9);
  cout<<"Performance : "<<flops<<" GFLOPS"<<endl;
  
  //synchronize host-gpu memory for file output
  hipDeviceSynchronize();
  
  //ouput result to .csv file
  ofstream fileOut;
  fileOut.open("cudaUnifiedMemHeatEq.csv");
  fileOut<<"x,y,z,T\n";
  for(int i=0; i<nx; ++i){
    for(int j=0; j<ny; ++j){
      int id = i*ny + j;
      float xg = (float)i*dx;
      float yg = (float)j*dy;
      fileOut<<setprecision(8);
      fileOut<<fixed;
      fileOut<<xg<<","
	     <<yg<<","
	     <<Told[id]<<","
	     <<Told[id]<<"\n";
    }
  }
  fileOut.close();


  //free memory
  hipFree(Told);
  hipFree(Tnew);

  return 0;
  
}
